#include "hip/hip_runtime.h"
#include "pairwise_hist.cuh"
#include "split_properties_helpers.cuh"
#include "compute_pair_hist_loop.cuh"
#include <hip/hip_cooperative_groups.h>
#include <catboost/cuda/cuda_lib/kernel/arch.cuh>
#include <catboost/cuda/cuda_util/kernel/instructions.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <cstdio>


using namespace cooperative_groups;

namespace NKernel {

    template<bool IsFullPass>
    struct TFiveBitPairwiseHistUnrollTrait {

        static constexpr int InnerUnroll() {
            #if __CUDA_ARCH__ <= 350
            return 4;
            #elif __CUDA_ARCH__ < 700
            return 2;
            #else
            return IsFullPass ? 4 : 8;
            #endif
        }

        static constexpr int OuterUnroll() {
            #if __CUDA_ARCH__ <= 350
            return 2;
            #elif __CUDA_ARCH__ < 700
            return 2;
            #else
            return 1;
            #endif
        }
    };



    template<int BlockSize, bool NeedLastBinMask /*is 32 histogram */, class TCmpBins = TCmpBinsWithoutOneHot>
    struct TFiveBitHistogram {
        TCmpBins CmpBinsFunc;
        float* Histogram;

        __forceinline__ __device__ int SliceOffset() {
            const int warpOffset = 1024 * (threadIdx.x / 32);
            //2 blocks if INNER_HIST_BITS_COUNT = 0, else 1
            // x4 feature and x4 histograms, though histStart = blockIdx * 16
            return warpOffset +  (threadIdx.x & 16);
        }


        __forceinline__  __device__ TFiveBitHistogram(float* buff, TCmpBins cmpBins)
        : CmpBinsFunc(cmpBins) {
            Histogram = buff;
            CmpBinsFunc = cmpBins;

            for (int i = threadIdx.x; i < BlockSize * 32; i += BlockSize) {
                Histogram[i] = 0;
            }
            Histogram += SliceOffset();
            __syncthreads();
        }



        __forceinline__ __device__ void AddPair(const ui32 ci1,
                                                const ui32 ci2,
                                                const float w) {
            thread_block_tile<16> groupTile = tiled_partition<16>(this_thread_block());

            const bool flag = threadIdx.x & 1;

            const int shift = 4 * (threadIdx.x & 6);
            const ui32 bins1 = RotateRight(flag ? ci2 : ci1, shift);
            const ui32 bins2 = RotateRight(flag ? ci1 : ci2, shift);

            #pragma unroll
            for (int i = 0; i < 4; i++) {
                const int f = (threadIdx.x + 2 * i) & 6;
                int bin1 = (bins1 >> (24 - 8 * i)) & 255;
                int bin2 = (bins2 >> (24 - 8 * i)) & 255;


                const float w1 = (!NeedLastBinMask || bin1 < 32) ? w : 0;
                const float w2 = (!NeedLastBinMask || bin2 < 32) ? w : 0;

                const int tmp = (CmpBinsFunc.Compare(i, bin1, bin2, flag) ? 0 : 8) + f;

                int offset1 = tmp + ((bin1 & 31) << 5) + flag;
                int offset2 = tmp + ((bin2 & 31) << 5) + !flag;


                groupTile.sync();

                if (groupTile.thread_rank() < 8) {
                    Histogram[offset1] += w1;
                }

                groupTile.sync();

                if (groupTile.thread_rank() >= 8) {
                    Histogram[offset1] += w1;
                }

                groupTile.sync();

                if (groupTile.thread_rank() < 8) {
                    Histogram[offset2] += w2;
                }

                groupTile.sync();

                if (groupTile.thread_rank() >= 8) {
                    Histogram[offset2] += w2;
                }
            }
        }


        #if __CUDA_ARCH__ < 700
        template <int N>
        __forceinline__ __device__ void AddPairs(const ui32* ci1,
                                                 const ui32* ci2,
                                                 const float* w) {
            #pragma unroll
            for (int k = 0; k < N; ++k) {
                AddPair(ci1[k], ci2[k], w[k]);
            }
        }
        #else
        template <int N>
        __forceinline__ __device__ void AddPairs(const ui32* ci1,
                                                 const ui32* ci2,
                                                 const float* w) {
            thread_block_tile<16> groupTile = tiled_partition<16>(this_thread_block());

            const bool flag = threadIdx.x & 1;
            const int shift = 4 * (threadIdx.x & 6);

            ui32 bins1[N];
            ui32 bins2[N];

            #pragma unroll
            for (int i = 0; i < 4; i++) {
                const int f = (threadIdx.x + 2 * i) & 6;

                int bin1[N];
                int bin2[N];

                float w1[N];
                float w2[N];

                int offset1[N];
                int offset2[N];

                #pragma unroll
                for (int k = 0; k < N;++k) {
                    if (i == 0) {
                        bins1[k] = RotateRight(flag ? ci2[k] : ci1[k], shift);
                        bins2[k] = RotateRight(flag ? ci1[k] : ci2[k], shift);
                    }
                    bin1[k] = (bins1[k] >> (24 - 8 * i)) & 255;
                    bin2[k] = (bins2[k] >> (24 - 8 * i)) & 255;

                    w1[k] = (!NeedLastBinMask || bin1[k] < 32) ? w[k] : 0;
                    w2[k] = (!NeedLastBinMask || bin2[k] < 32) ? w[k] : 0;

                    const int tmp = (CmpBinsFunc.Compare(i, bin1[k], bin2[k], flag)  ? 0 : 8) + f;
                    offset1[k] = tmp + ((bin1[k] & 31) * 32) + flag;
                    offset2[k] = tmp + ((bin2[k] & 31) * 32) + !flag;
                }


                groupTile.sync();

                if (groupTile.thread_rank() < 8) {
                    #pragma unroll
                    for (int k = 0; k < N; ++k) {
                        Histogram[offset1[k]] += w1[k];
                    }
                }

                groupTile.sync();

                if (groupTile.thread_rank() >= 8) {
                    #pragma unroll
                    for (int k = 0; k < N; ++k) {
                        Histogram[offset1[k]] += w1[k];
                    }
                }

                groupTile.sync();

                if (groupTile.thread_rank() < 8) {
                    #pragma unroll
                    for (int k = 0; k < N; ++k) {
                        Histogram[offset2[k]] += w2[k];
                    }
                }

                groupTile.sync();

                if (groupTile.thread_rank() >= 8) {
                    #pragma unroll
                    for (int k = 0; k < N; ++k) {
                        Histogram[offset2[k]] += w2[k];
                    }
                }
            }
        }
        #endif

        __forceinline__ __device__  void Reduce() {
            Histogram -= SliceOffset();
            __syncthreads();

            {
                const int warpHistSize = 1024;

                for (int start = threadIdx.x; start < warpHistSize; start += BlockSize) {
                    float sum = 0;

                    #pragma unroll 12
                    for (int i = start; i < 32 * BlockSize; i += warpHistSize) {
                        sum += Histogram[i];
                    }

                    Histogram[warpHistSize + start] = sum;
                }
            }
            __syncthreads();

            const int maxFoldCount = 32;
            const int fold = (threadIdx.x >> 1) & 31;
            const int f = threadIdx.x / 64;


            if (threadIdx.x < 256) {
                float weightLeq = 0;
                float weightGe = 0;
                const bool isSecondBin = (threadIdx.x & 1);

                if (fold < maxFoldCount) {
                    const volatile float* __restrict__ src = Histogram
                                                             + 1024  //warpHistSize
                                                             + 32 * fold
                                                             + 2 * f
                                                             + isSecondBin;

                    weightLeq = src[0] + src[16];
                    weightGe = src[8] + src[24];

                    Histogram[4 * (maxFoldCount * f + fold) + isSecondBin] = weightLeq;
                    Histogram[4 * (maxFoldCount * f + fold) + 2 + isSecondBin] = weightGe;
                }
            }

            __syncthreads();
        }
    };


    template<int BlockSize, bool IsFullPass, int M>
    #if __CUDA_ARCH__ <= 350
    __launch_bounds__(BlockSize, 1)
    #elif __CUDA_ARCH__ < 700
    __launch_bounds__(BlockSize, 2)
    #endif
    __global__ void ComputeSplitPropertiesNonBinaryPairs(const TCFeature* feature, int fCount, const ui32* cindex,
                                                         const uint2* pairs, const float* weight,
                                                         const TDataPartition* partition,
                                                         int histLineSize,
                                                         float* histogram) {

        const int featureOffset = (blockIdx.x / M) * 4;
        feature += featureOffset;
        cindex += feature->Offset;
        fCount = min(fCount - featureOffset, 4);


        __shared__ float localHist[32 * BlockSize];

        const int maxBinCount = GetMaxBinCount(feature, fCount, (int*) &localHist[0]);

        if (maxBinCount > 32) {
            return;
        }
        __syncthreads();


        if (IsFullPass) {
            partition += blockIdx.y;
            histogram += blockIdx.y * histLineSize * 4ULL;
        } else {
            const int depth = (int)log2((float)gridDim.y);
            int partId = GetPairwisePartIdToCalculate(partition);
            partition += partId;
            histogram += (((blockIdx.z + 1) << depth) | blockIdx.y) * histLineSize * 4ULL;
        }

        if (partition->Size == 0) {
            return;
        }


        constexpr int histBlockCount = 1;
        constexpr int innerUnroll = TFiveBitPairwiseHistUnrollTrait<IsFullPass>::InnerUnroll();
        constexpr int outerUnroll = TFiveBitPairwiseHistUnrollTrait<IsFullPass>::OuterUnroll();

        const bool needOneHot = HasOneHotFeatures(feature, fCount, (int*)&localHist[0]);

        #define DECLARE_PASS(NEED_MASK, TBinCmp)   \
        {                                   \
            using THist = TFiveBitHistogram<BlockSize, NEED_MASK, TBinCmp>;\
            TBinCmp cmp(feature, fCount);\
            THist hist(&localHist[0], cmp);\
            ComputePairHistogram< BlockSize, histBlockCount, innerUnroll, outerUnroll, M, THist>(partition->Offset, cindex, partition->Size, pairs, weight, hist);\
        }
        if (maxBinCount < 32) {
            if (needOneHot) {
                DECLARE_PASS(false, TCmpBinsWithOneHot<4>);
            } else {
                DECLARE_PASS(false, TCmpBinsWithoutOneHot);
            }
        } else {
            if (needOneHot) {
                DECLARE_PASS(true, TCmpBinsWithOneHot<4>);
            } else {
                DECLARE_PASS(true, TCmpBinsWithoutOneHot);
            }
        }
        #undef DECLARE_PASS

        if (threadIdx.x < 256) {
            const int histId = threadIdx.x & 3;
            const int binId = (threadIdx.x >> 2) & 15;
            const int fid = (threadIdx.x >> 6) & 3;
            const int maxFoldCount = 1 << 5;

            if (fid < fCount) {
                const ui32 bfStart = feature[fid].FirstFoldIndex;
                histogram += 4 * bfStart;

                for (int fold = binId; fold < feature[fid].Folds; fold += 16) {
                    const int readOffset = 4 * (maxFoldCount * fid + fold) + histId;
                    if (M > 1) {
                        atomicAdd(histogram + 4 * fold + histId, localHist[readOffset]);
                    } else {
                        histogram[4 * fold + histId] += localHist[readOffset];
                    }
                }
            }
        }
    }



    void ComputePairwiseHistogramOneByte5Bits(const TCFeature* features,
                                              const ui32 featureCount,
                                              const ui32 fiveBitsFeatureCount,
                                              const ui32* compressedIndex,
                                              const uint2* pairs, ui32 pairCount,
                                              const float* weight,
                                              const TDataPartition* partition,
                                              ui32 partCount,
                                              ui32 histLineSize,
                                              bool fullPass,
                                              float* histogram,
                                              TCudaStream stream) {

        if (fiveBitsFeatureCount > 0) {
            const int blockSize = 384;
            dim3 numBlocks;
            numBlocks.x = (fiveBitsFeatureCount+ 3) / 4;
            numBlocks.y = fullPass ? partCount : partCount / 4;
            numBlocks.z = fullPass ? 1 : 3;
            const ui32 blockPerFeatureMultiplier = EstimateBlockPerFeatureMultiplier(numBlocks, pairCount, 64);
            numBlocks.x = (featureCount + 3) / 4;
            numBlocks.x *= blockPerFeatureMultiplier;



            #define NB_HIST(IS_FULL, BLOCKS_PER_FEATURE)   \
            ComputeSplitPropertiesNonBinaryPairs < blockSize, IS_FULL, BLOCKS_PER_FEATURE > << <numBlocks, blockSize, 0, stream>>>(\
                                                  features, featureCount, compressedIndex,  pairs,\
                                                  weight, partition,  histLineSize, histogram);

            #define DISPATCH(BLOCKS_PER_FEATURE)  \
            if (fullPass) {                       \
                NB_HIST(true, BLOCKS_PER_FEATURE) \
            } else {                              \
                NB_HIST(false, BLOCKS_PER_FEATURE)\
            }


            if (blockPerFeatureMultiplier == 1) {
                DISPATCH(1);
            } else if (blockPerFeatureMultiplier == 2) {
                DISPATCH(2);
            } else if (blockPerFeatureMultiplier == 4) {
                DISPATCH(4);
            } else if (blockPerFeatureMultiplier == 8) {
                DISPATCH(8);
            } else if (blockPerFeatureMultiplier == 16) {
                DISPATCH(16);
            } else if (blockPerFeatureMultiplier == 32) {
                DISPATCH(32);
            } else if (blockPerFeatureMultiplier == 64) {
                DISPATCH(64);
            } else {
                exit(0);
            }
            #undef NB_HIST
            #undef DISPATCH
        }
    }
}
