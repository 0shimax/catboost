#include "hip/hip_runtime.h"
#include "random.cuh"
#include "random_gen.cuh"
#include "catboost/cuda/cuda_lib/kernel/arch.cuh"

namespace NKernel {

    __global__ void PoissonRandImpl(ui64* seeds, uint seedSize,
                                    const float* alpha, int* result)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextPoisson(&s, alpha[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void PoissonRand(ui64* seeds, uint size, const float* alphas, int* result, TCudaStream stream)
    {
        const uint blockSize = 256;
        const uint numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        PoissonRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, alphas, result);
    }

    __global__ void GaussianRandImpl(ui64* seeds, uint seedSize, float* result)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextNormal(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GaussianRand(ui64* seeds, uint size, float* result, TCudaStream stream)
    {
        const uint blockSize = 256;
        const uint numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GaussianRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void UniformRandImpl(ui64* seeds, uint seedSize, float* result)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextUniform(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void UniformRand(ui64* seeds, uint size, float* result, TCudaStream stream)
    {
        const uint blockSize = 256;
        const uint numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        UniformRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void GammaRandImpl(ui64* seeds, const float* alphas,
                                  const float* scale, uint seedSize, float* result)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextGamma(&s, alphas[i], scale[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GammaRand(ui64* seeds, const float* alphas, const float* scale,
                   uint size, float* result, TCudaStream stream)
    {
        const uint blockSize = 256;
        const uint numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GammaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, scale, size, result);
    }

    __global__ void BetaRandImpl(ui64* seeds, const float* alphas,
                                 const float* betas, uint seedSize, float* result)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextBeta(&s, alphas[i], betas[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void BetaRand(ui64* seeds, const float* alphas, const float* betas,
                  uint size, float* result, TCudaStream stream)
    {
        const uint blockSize = 256;
        const uint numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        BetaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, betas, size, result);
    }

}
