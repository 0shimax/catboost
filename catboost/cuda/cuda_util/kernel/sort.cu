#include "sort.cuh"
#include <contrib/libs/cub/cub/device/device_radix_sort.cuh>
#include <cassert>

namespace NKernel {

    template <typename K, typename V> hipError_t RadixSort(K* keys, V *values, ui32 size, TRadixSortContext& context, TCudaStream stream)
    {
        hipcub::DoubleBuffer<K> doubleBufferKeys(keys, context.GetTempKeys<K>());
        hipError_t error;

        if (context.ValueSize) {
            hipcub::DoubleBuffer<V> doubleBufferValues(values, context.GetTempValues<V>());

            if (context.Descending) {


                hipcub::DoubleBuffer<K> inputValues;
                error = hipcub::DeviceRadixSort::SortPairsDescending(context.TempStorage, context.TempStorageSize,
                                                                 doubleBufferKeys,
                                                                 doubleBufferValues,
                                                                 size,
                                                                 context.FirstBit, context.LastBit,
                                                                 stream);
            } else {
                error = hipcub::DeviceRadixSort::SortPairs(context.TempStorage, context.TempStorageSize,
                                                       doubleBufferKeys,
                                                       doubleBufferValues,
                                                       size,
                                                       context.FirstBit, context.LastBit,
                                                       stream);
            }
            if (doubleBufferValues.Current() != values) {
                assert(sizeof(V) == context.ValueSize);
                hipMemcpyAsync(values, doubleBufferValues.Current(), sizeof(V) * size, hipMemcpyDefault, stream);
            }
        } else {
            if (context.Descending) {
                 error = hipcub::DeviceRadixSort::SortKeysDescending(context.TempStorage, context.TempStorageSize,
                                                                doubleBufferKeys,
                                                                size,
                                                                context.FirstBit, context.LastBit,
                                                                stream);
                } else {
                    error = hipcub::DeviceRadixSort::SortKeys(context.TempStorage, context.TempStorageSize,
                                                           doubleBufferKeys,
                                                           size,
                                                           context.FirstBit, context.LastBit,
                                                           stream);
                }
        }
        //TODO(noxoomo): error handling
        if (doubleBufferKeys.Current() != keys) {
            hipMemcpyAsync(keys, doubleBufferKeys.Current(), sizeof(K) * size, hipMemcpyDefault, stream);
        }
        return error;
    }

    template hipError_t RadixSort(uchar* keys, uchar* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(uchar* keys, ushort* values, uint size,TRadixSortContext& context,  TCudaStream stream);
    template hipError_t RadixSort(uchar* keys, uint* values, uint size, TRadixSortContext& context,  TCudaStream stream);

    template hipError_t RadixSort(ushort* keys, uchar* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(ushort* keys, ushort* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(ushort* keys, uint* values, uint size, TRadixSortContext& context, TCudaStream stream);

    template hipError_t RadixSort(uint* keys, uchar* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(uint* keys, ushort* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(uint* keys, uint* values, uint size, TRadixSortContext& context,  TCudaStream stream);

    template hipError_t RadixSort(float* keys, uchar* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(float* keys, ushort* values, uint size, TRadixSortContext& context, TCudaStream stream);
    template hipError_t RadixSort(float* keys, uint* values, uint size, TRadixSortContext& context,  TCudaStream stream);



}
